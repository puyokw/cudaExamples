
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void helloFromGPU(){
	printf("Hello World from GPU!\n");
}

int main(){
	printf("Hello World from CPU!\n");

	helloFromGPU <<< 1,10>>>();
	hipDeviceReset();
}