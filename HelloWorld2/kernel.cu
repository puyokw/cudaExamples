
#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void helloFromGPU(){
	printf("Hello World from GPU\n");
}

extern "C"
void launch_helloFromGPU(){
	helloFromGPU<<< 1, 10 >>>();
	hipDeviceReset();
}